#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <assert.h>
#include <time.h>
#include <iostream>
#include <fstream>
using namespace std;
// CUDA runtime
#include <hip/hip_runtime.h>
// helper functions and utilities to work with CUDA
#include <>

typedef unsigned char u8;
typedef unsigned int u32;
typedef unsigned long long u64;

#define N 38
#define count 274877906944
#define THREAD_NUM 1024
#define BLOCK_NUM 2048
#define roundnum 844

__device__ u8 Trivium(u8* gpu_key, u8* gpu_pos, u64 num, u8* gpu_Free_Cube, u8 iv_loc, u8 freecubedim)
{
    
    u32 value[N] = { 0 };
    u32 IV[3] = { 0,0,0 };
    u32 i;
    for (i = 0; i < N; i++)
        value[i] = ((num >> i) & 0x01);
    for (i = 0; i < N; i++)
        if (value[i] == 1)
            IV[(gpu_pos[i] >> 5)] |= value[i] << (gpu_pos[i] & 0x1f);

    for (i = 0; i < freecubedim; i++)
    {
        if (iv_loc & 1)
        {
            if (gpu_Free_Cube[i] < 32)
            {
                IV[0] = IV[0] ^ (1 << gpu_Free_Cube[i]);
            }
            else if (gpu_Free_Cube[i] < 64)
            {
                IV[1] = IV[1] ^ (1 << (gpu_Free_Cube[i] - 32));
            }
            else if (gpu_Free_Cube[i] < 80)
            {
                IV[2] = IV[2] ^ (1 << (gpu_Free_Cube[i] - 64));
            }
        }
        iv_loc = iv_loc >> 1;
    }

    



    u32 t1, t2, t3;
    u32 s0, s1, s2, s3, s4, s5, s6, s7, s8, s9;
    u32 temp1, temp2, temp3, temp4, temp5, temp6;
    u32 z, z1;

    s0 = gpu_key[0] ^ (gpu_key[1] << 8) ^ (gpu_key[2] << 16) ^ (gpu_key[3] << 24);
    s1 = gpu_key[4] ^ (gpu_key[5] << 8) ^ (gpu_key[6] << 16) ^ (gpu_key[7] << 24);
    s2 = gpu_key[8] ^ (gpu_key[9] << 8);
    s3 = IV[0];
    s4 = IV[1];
    s5 = IV[2];
    s6 = 0;
    s7 = 0;
    s8 = 0;
    s9 = 0x00007000;

    u32 roundnum_word = roundnum / 32;
    u32 roundnum_bit = roundnum % 32;
    for (i = 0; i < roundnum_word; i++)
    {
        temp1 = (s2 << 30) | (s1 >> 2);
        temp2 = (s2 << 3) | (s1 >> 29);
        temp3 = (s5 << 27) | (s4 >> 5);
        temp4 = (s5 << 12) | (s4 >> 20);
        temp5 = (s8 << 30) | (s7 >> 2);
        temp6 = (s9 << 17) | (s8 >> 15);

        t1 = temp1 ^ temp2;
        t2 = temp3 ^ temp4;
        t3 = temp5 ^ temp6;
        //z = t1^t2^t3;

        //  t1 = t1 + s91s92 + s171
        temp1 = (s2 << 5) | (s1 >> 27);
        temp2 = (s2 << 4) | (s1 >> 28);
        temp3 = (s5 << 18) | (s4 >> 14);
        t1 ^= (temp1 & temp2) ^ temp3;

        //  t2 = t2 + s175s176 + s264
        temp1 = (s5 << 14) | (s4 >> 18);
        temp2 = (s5 << 13) | (s4 >> 19);
        temp3 = (s8 << 9) | (s7 >> 23);
        t2 ^= (temp1 & temp2) ^ temp3;

        //  t3 = t3 + s286s287 + s69
        temp1 = (s9 << 19) | (s8 >> 13);
        temp2 = (s9 << 18) | (s8 >> 14);
        temp3 = (s2 << 27) | (s1 >> 5);
        t3 ^= (temp1 & temp2) ^ temp3;

        // update register 1
        s2 = (s1) & (0x1FFFFFFF);
        s1 = s0;
        s0 = t3;

        //  update register 2
        s5 = s4 & (0x000FFFFF);
        s4 = s3;
        s3 = t1;

        //  update register 3
        s9 = s8 & (0x00007FFF);
        s8 = s7;
        s7 = s6;
        s6 = t2;
    }
    if (roundnum_bit != 0)
    {
        temp1 = (s2 << 30) | (s1 >> 2);//���֮���൱��ֱ�Ӽ���
        temp2 = (s2 << 3) | (s1 >> 29);
        temp3 = (s5 << 27) | (s4 >> 5);
        temp4 = (s5 << 12) | (s4 >> 20);
        temp5 = (s8 << 30) | (s7 >> 2);
        temp6 = (s9 << 17) | (s8 >> 15);

        t1 = temp1 ^ temp2;
        t2 = temp3 ^ temp4;
        t3 = temp5 ^ temp6;
        z1 = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;
        //z1=z;

        //t1 = t1 + s91s92 + s171
        temp1 = (s2 << 5) | (s1 >> 27);
        temp2 = (s2 << 4) | (s1 >> 28);
        temp3 = (s5 << 18) | (s4 >> 14);

        t1 ^= (temp1 & temp2) ^ temp3;

        //t2 = t2 + s175s176 + s264
        temp1 = (s5 << 14) | (s4 >> 18);
        temp2 = (s5 << 13) | (s4 >> 19);
        temp3 = (s8 << 9) | (s7 >> 23);

        t2 ^= (temp1 & temp2) ^ temp3;

        //t3 = t3 + s286s287 + s69
        temp1 = (s9 << 19) | (s8 >> 13);
        temp2 = (s9 << 18) | (s8 >> 14);
        temp3 = (s2 << 27) | (s1 >> 5);

        t3 ^= (temp1 & temp2) ^ temp3;

        // update register 1
        s2 = (s1) & (0x1FFFFFFF);
        s1 = s0;
        s0 = t3;

        //update register 2
        s5 = s4 & (0x000FFFFF);
        s4 = s3;
        s3 = t1;

        //update register 3
        s9 = s8 & (0x00007FFF);
        s8 = s7;
        s7 = s6;
        s6 = t2;
    }
    temp1 = (s2 << 30) | (s1 >> 2);
    temp2 = (s2 << 3) | (s1 >> 29);
    temp3 = (s5 << 27) | (s4 >> 5);
    temp4 = (s5 << 12) | (s4 >> 20);
    temp5 = (s8 << 30) | (s7 >> 2);
    temp6 = (s9 << 17) | (s8 >> 15);
    z = temp1 ^ temp2 ^ temp3 ^ temp4 ^ temp5 ^ temp6;
    if (roundnum_bit != 0)
        z = (z1 << roundnum_bit) | (z >> (32 - roundnum_bit));
    u8 tmp = z >> 31;
    return tmp;
}

__global__ void kernel(u8* gpu_truthtable, u8* gpu_key, u8* gpu_pos, u8* gpu_Free_Cube, u8 iv_loc, u8 freecubedim)
{

    const  u32 tidx = threadIdx.x;
    const  u32 bid = blockIdx.x;
    u8 result = 0;
    for (u64 i = bid * THREAD_NUM + tidx; i < count; i = i + BLOCK_NUM * THREAD_NUM)
    {
        result ^= Trivium(gpu_key, gpu_pos, i, gpu_Free_Cube, iv_loc, freecubedim);
    }
    gpu_truthtable[bid * THREAD_NUM + tidx] = result;
}

u8 excute(u8* Key, u8* Pos, u8* Free_Cube, u8 iv_loc, u8 freecubedim)
{
    u8* gpu_key, * gpu_pos, * gpu_truthtable, * gpu_Free_Cube;
    u8* truthtable = (u8*)calloc(THREAD_NUM * BLOCK_NUM, sizeof(u8));


    hipMalloc((void**)&gpu_truthtable, THREAD_NUM * BLOCK_NUM * sizeof(u8));
    hipMalloc((void**)&gpu_key, 10 * sizeof(u8));
    hipMalloc((void**)&gpu_pos, N * sizeof(u8));
    hipMalloc((void**)&gpu_Free_Cube, freecubedim * sizeof(u8));

    hipMemcpy(gpu_key, Key, 10 * sizeof(u8), hipMemcpyHostToDevice);
    hipMemcpy(gpu_pos, Pos, N * sizeof(u8), hipMemcpyHostToDevice);
    hipMemcpy(gpu_Free_Cube, Free_Cube, freecubedim * sizeof(u8), hipMemcpyHostToDevice);

    kernel << <BLOCK_NUM, THREAD_NUM >> > (gpu_truthtable, gpu_key, gpu_pos, gpu_Free_Cube, iv_loc, freecubedim);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess)
    {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(truthtable, gpu_truthtable, THREAD_NUM * BLOCK_NUM * sizeof(u8), hipMemcpyDeviceToHost);

    u8 result = 0;
    for (u64 i = 0; i < THREAD_NUM * BLOCK_NUM; i++)
    {
        result = result ^ truthtable[i];
    }
    
    free(truthtable);
    hipFree(gpu_truthtable);
    hipFree(gpu_key);
    hipFree(gpu_pos);
    hipFree(gpu_Free_Cube);


    hipDeviceReset();
    return result;
}


int qiumi(int x, int y)
{
    int result = 1;
    for (size_t i = 0; i < y; i++)
    {
        result = result * x;
    }
    return result;
}



int main()
{

    u8 cube[21][38] = {
        {0,2,4,6,8,9,11,13,15,17,19,21,23,26,28,30,32,34,36,38,41,43,45,47,49,51,53,56,58,60,62,64,66,68,71,73,75,79},
        {0,1,2,4,6,8,11,13,15,17,19,21,23,26,28,30,32,34,36,38,41,43,45,47,49,51,53,56,58,60,62,64,66,68,71,73,75,79},
        {1,3,5,7,8,10,12,14,16,18,20,22,25,27,29,31,33,35,37,40,42,44,46,48,50,52,55,57,59,61,63,65,67,70,72,74,76,78},
        {0,2,4,6,9,11,13,15,17,19,21,24,26,28,30,32,34,36,39,41,43,45,47,49,51,54,56,58,60,62,64,66,69,71,73,75,77,79},
        {0,2,4,6,8,10,12,15,17,19,21,23,25,27,30,32,34,36,38,40,42,45,47,49,51,53,55,57,60,62,64,66,68,70,72,75,77,79},
        {0,1,2,4,7,9,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,79},
        {0,2,4,6,9,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,77,79},
        {0,1,2,4,6,9,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,79},
        {1,3,5,7,9,10,12,14,16,18,20,22,25,27,29,31,33,35,37,40,42,44,46,48,50,52,55,57,59,61,63,65,67,70,72,74,76,78},
        {0,2,4,7,9,10,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,79},
        {0,1,3,5,7,9,11,13,16,18,20,22,24,26,28,31,33,35,37,39,41,43,46,48,50,52,54,56,58,61,63,65,67,69,71,73,76,78},
        {0,2,4,6,7,9,11,13,15,17,19,21,24,26,28,30,32,34,36,39,41,43,45,47,49,51,54,56,58,60,62,64,66,69,71,73,75,79},
        {1,3,5,6,7,10,12,14,16,18,20,22,25,27,29,31,33,35,37,40,42,44,46,48,50,52,55,57,59,61,63,65,67,70,72,74,76,78},
        {0,1,2,4,6,9,11,13,15,17,19,21,24,26,28,30,32,34,36,39,41,43,45,47,49,51,54,56,58,60,62,64,66,69,71,73,75,79},
        {0,2,4,6,8,10,11,13,15,17,19,21,23,26,28,30,32,34,36,38,41,43,45,47,49,51,53,56,58,60,62,64,66,68,71,73,75,79},
        {0,2,4,7,8,9,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,79},
        {0,2,4,6,8,9,11,13,15,17,19,21,24,26,28,30,32,34,36,39,41,43,45,47,49,51,54,56,58,60,62,64,66,69,71,73,75,79},
        {0,2,4,6,9,10,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,79},
        {0,2,4,7,9,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,77,79},
        {1,3,5,7,8,9,11,13,16,18,20,22,24,26,28,31,33,35,37,39,41,43,46,48,50,52,54,56,58,61,63,65,67,69,71,73,76,78},
        {0,2,4,8,9,10,11,13,15,17,19,22,24,26,28,30,32,34,37,39,41,43,45,47,49,52,54,56,58,60,62,64,67,69,71,73,75,79},
};

    u8 Free_Cube[21][4] = {
        {1,10,76,77},
        {9,10,76,77},
        {0,2,9,11},
        {1,7,8,10},
        {3,5,7},
        {6,8,10,77},
        {7,8,10,20},
        {8,10,20,77},
        {2,8,11},
        {1,5,6,8},
        {2,6,79},
        {1,5,8,10},
        {0,8,9,11},
        {8,10,76,77},
        {1,9,76,77},
        {6,10,77},
        {1,7,10,76},
        {1,7,8,20},
        {1,5,6,8},
        {0,6,79},
        {5,6,7,77},
};

    u8 freecubedim[21] = {4,4,4,4,3,4,4,4,3,4,3,4,4,4,4,3,4,4,4,3,4};


    clock_t start_time, finish_time;
    u8 key[10] = { 0 };
    u8 result, i, j;
    u8 iv_loc;
    srand((unsigned)time(NULL)); 
    u32 flag = 0;

    for (i = 0; i < 128; i++)
    {
        for (j = 0; j < 10; j++)
        { 
            key[j] += rand() % 256;
            printf("%d ", key[j]);
        } 
        printf("\n");

        start_time = clock();
        for (j = 0; j < 21; j++)
        {
            for (iv_loc = 0; iv_loc < qiumi(2,freecubedim[j]); iv_loc++)
            {
                flag++;
                result = excute(key, cube[j], Free_Cube[j], iv_loc, freecubedim[j]);
                if (result == 1)
                    break;
            }
            printf("%d,", result);
        }
        printf("\n");
        finish_time = clock();
        printf("IV:%d, time:%f\n", flag, (float)(finish_time - start_time) / CLOCKS_PER_SEC);
    }

    return 0;
}
 
